#include <stdio.h>

#ifdef EASTNN_USE_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include"nncuda.h"

namespace easynn{
    


void* fastCudaMalloc(size_t size)
{
    void *ptr = 0;
    CHECK(hipMalloc((void **)&ptr, size + EASYNN_MALLOC_OVERREAD));
    return ptr;
}

void fastCudaFree(void * ptr)
{
    CHECK(hipFree(ptr));
}


#endif   //EASTNN_USE_CUDA



}//namespace