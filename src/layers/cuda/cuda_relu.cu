#include "hip/hip_runtime.h"
#include"nncuda.h"
#include"cuda_relu.h"
#include<stdio.h>


namespace easynn{


__global__ void cuda_relu_forward(float* input,float* output,int n)
{
    CUDA_KERNEL_LOOP(index, n) 
    {   
        output[index] = fmaxf(0.0f,input[index]);  
    }

}



void cuda_relu(const Mat& input,Mat& output,const Optional& op)
{
    int count=input.w*input.h*input.c*input.d;

    int nbytes = count*input.elemsize;

    Mat cu_mat_input = input.reshape(count);
    Mat cu_mat_output(count);

    float* d_input,*d_output;

    CHECK(hipMalloc(&d_input,nbytes));
    CHECK(hipMalloc(&d_output,nbytes));

    hipMemcpy(d_input, cu_mat_input.data, nbytes, hipMemcpyHostToDevice);

    cuda_relu_forward<<<CAFFE_GET_BLOCKS(count), EASYNN_CUDA_NUM_THREADS>>>(d_input,d_output,count);

    hipMemcpy(cu_mat_output.data, d_output, nbytes, hipMemcpyDeviceToHost);


    if (input.dims == 1)
        output=cu_mat_output.reshape(input.w);
    else if (input.dims == 2)
        output=cu_mat_output.reshape(input.w, input.h);
    else if (input.dims == 3)
        output=cu_mat_output.reshape(input.w, input.h, input.c);
    else if (input.dims == 4)
        output=cu_mat_output.reshape(input.w, input.h, input.d, input.c);

    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
        
}

}// namespace