#include "hip/hip_runtime.h"
#include"nncuda.h"
#include"cuda_relu.h"
#include<stdio.h>


namespace easynn{


__global__ void cuda_relu_forward(float* input,float* output,int n)
{
    CUDA_KERNEL_LOOP(index, n) 
    {   
        output[index] = fmaxf(0.0f,input[index]);  
    }

}

void cuda_relu(const Mat& input,Mat& output,const Optional& op)
{
    int count=input.w*input.h*input.c*input.d;

    int nbytes = count*input.elemsize;

    Mat cu_mat_input = input.reshape(count);
    Mat cu_mat_output(count);

    float* d_input,*d_output;

    CHECK(hipMalloc(&d_input,nbytes));
    CHECK(hipMalloc(&d_output,nbytes));

    hipMemcpy(d_input, cu_mat_input.data, nbytes, hipMemcpyHostToDevice);

    cuda_relu_forward<<<EASYNN_GET_BLOCKS(count), EASYNN_CUDA_NUM_THREADS>>>(d_input,d_output,count);

    hipMemcpy(cu_mat_output.data, d_output, nbytes, hipMemcpyDeviceToHost);


    if (input.dims == 1)
        output=cu_mat_output.reshape(input.w);
    else if (input.dims == 2)
        output=cu_mat_output.reshape(input.w, input.h);
    else if (input.dims == 3)
        output=cu_mat_output.reshape(input.w, input.h, input.c);
    else if (input.dims == 4)
        output=cu_mat_output.reshape(input.w, input.h, input.d, input.c);

    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
        
}


__global__ void cuda_relu_vec_forward(float *input,float*output,size_t n)
{

    int gtid =blockIdx.x*blockDim.x+threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for(int index=gtid;index<(n+CUDA_VEC_SIZE-1)/CUDA_VEC_SIZE;index+=stride){
        float4 a = reinterpret_cast<float4*>(input)[index];
        float4 c;
        c.x = fmaxf(0.0f,a.x); 
        c.y = fmaxf(0.0f,a.y);
        c.z = fmaxf(0.0f,a.z);
        c.w = fmaxf(0.0f,a.w);
        reinterpret_cast<float4*>(output)[index] = c;
    }

}

void cuda_relu_vec(const Mat& input,Mat& output,const Optional& op)
{
    int count=input.w*input.h*input.c*input.d;

    int N = (count+CUDA_VEC_SIZE-1)/CUDA_VEC_SIZE*CUDA_VEC_SIZE;

    int nbytes = N*input.elemsize;

    Mat cu_mat_input = input.reshape(count);
    Mat cu_mat_output(count);

    float* d_input,*d_output;

    CHECK(hipMalloc(&d_input,nbytes));
    CHECK(hipMalloc(&d_output,nbytes));
    hipMemset(d_input, 0, nbytes);


    hipMemcpy(d_input, cu_mat_input.data, count*sizeof(float), hipMemcpyHostToDevice);

    cuda_relu_vec_forward<<<EASYNN_GET_VEC_BLOCKS(count), EASYNN_CUDA_NUM_THREADS>>>(d_input,d_output,count);

    hipMemcpy(cu_mat_output.data, d_output, count*sizeof(float), hipMemcpyDeviceToHost);


    if (input.dims == 1)
        output=cu_mat_output.reshape(input.w);
    else if (input.dims == 2)
        output=cu_mat_output.reshape(input.w, input.h);
    else if (input.dims == 3)
        output=cu_mat_output.reshape(input.w, input.h, input.c);
    else if (input.dims == 4)
        output=cu_mat_output.reshape(input.w, input.h, input.d, input.c);

    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
        
}

}// namespace