#include "hip/hip_runtime.h"
#include"nncuda.h"
#include"cuda_silu.h"
#include<stdio.h>


namespace easynn{


__global__ void cuda_silu_forward(float* input,float* output,int n)
{
    CUDA_KERNEL_LOOP(index, n) 
    {   
        //printf("%f     %f\n",input[index],1.0f / (1.0f + expf(input[index])));
        output[index] = input[index] / (1.0f + expf(-input[index]));  
    }

}



void cuda_silu(const Mat& input,Mat& output,const Optional& op)
{
    int count=input.w*input.h*input.c*input.d;

    int nbytes = count*input.elemsize;

    Mat cu_mat_input = input.reshape(count);
    Mat cu_mat_output(count);

    float* d_input  =  (float*)fastCudaMalloc(nbytes);
    float* d_output =  (float*)fastCudaMalloc(nbytes);

    hipMemcpy(d_input, cu_mat_input.data, nbytes, hipMemcpyHostToDevice);

    cuda_silu_forward<<<CAFFE_GET_BLOCKS(count), EASYNN_CUDA_NUM_THREADS>>>(d_input,d_output,count);

    hipMemcpy(cu_mat_output.data, d_output, nbytes, hipMemcpyDeviceToHost);


    if (input.dims == 1)
        output=cu_mat_output.reshape(input.w);
    else if (input.dims == 2)
        output=cu_mat_output.reshape(input.w, input.h);
    else if (input.dims == 3)
        output=cu_mat_output.reshape(input.w, input.h, input.c);
    else if (input.dims == 4)
        output=cu_mat_output.reshape(input.w, input.h, input.d, input.c);

    fastCudaFree(d_input);
    fastCudaFree(d_output);
        
}

}// namespace